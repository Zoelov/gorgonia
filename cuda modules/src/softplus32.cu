
#include <hip/hip_runtime.h>
#include <math.h>

#ifdef __cplusplus
extern "C" {
#endif


__global__ void softplus32(float* A, int size)
{
	int idx = threadIdx.x;
	if (idx >= size) {
		return;
	}
	A[idx] = log1p(exp(A[idx]));
}
	
#ifdef __cplusplus
}
#endif